#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

__device__ int foundFlag = 0;
#define NUM_BLOCKS 256
#define NUM_THREADS 512

__global__ void findNonce(BYTE *block_hash, uint64_t *nonce, size_t *current_length, BYTE *block_content) {
	uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0) index ++;
	uint64_t stride = blockDim.x * gridDim.x;

	char nonce_string[NONCE_SIZE];
	BYTE gpu_block_hash[SHA256_HASH_SIZE];
	BYTE gpu_block_content[BLOCK_SIZE];
	BYTE my_diff[SHA256_HASH_SIZE] = "0000099999999999999999999999999999999999999999999999999999999999";
	memcpy(gpu_block_content, block_content, BLOCK_SIZE * sizeof(BYTE));

  	for (u_int64_t i = index; i <= MAX_NONCE; i+= stride) {
		if (foundFlag == 0) {
			int length = intToString(i, nonce_string);
        	d_strcpy((char*) gpu_block_content + *current_length, nonce_string);
        	apply_sha256(gpu_block_content, d_strlen((const char*)gpu_block_content), gpu_block_hash, 1);

        	if (compare_hashes(gpu_block_hash, my_diff) <= 0) {
				atomicExch(&foundFlag, 1);
				*nonce = i;
				memcpy(block_hash, gpu_block_hash, SHA256_HASH_SIZE * sizeof(BYTE));
            	break;
        	}
		} else {
			break;
		}
	}

	if (foundFlag != 0) {
		return;
	}

	return;
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE];

	BYTE *block_hash;
	uint64_t *nonce;
	size_t *current_length;
	BYTE *block_content;
	hipMallocManaged(&block_hash, SHA256_BLOCK_SIZE * sizeof(BYTE));
  	hipMallocManaged(&nonce, sizeof(uint64_t));
	hipMallocManaged(&current_length, sizeof(size_t));
	hipMallocManaged(&block_content, BLOCK_SIZE * sizeof(BYTE));

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	*current_length = strlen((char*) block_content);

	hipEvent_t start, stop;
	startTiming(&start, &stop);

	findNonce<<<NUM_BLOCKS, NUM_THREADS>>>(block_hash, nonce, current_length, block_content);
	hipDeviceSynchronize();

	float seconds = stopTiming(&start, &stop);
	printResult(block_hash, *nonce, seconds);

	hipFree(block_hash);
  	hipFree(nonce);
  	hipFree(current_length);
  	hipFree(block_content);

	return 0;
}
